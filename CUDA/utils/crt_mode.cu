#include "hip/hip_runtime.h"
#include "crt_mode.h"
#include "assert.h"

/*
void update_iv(unsigned char* iv[16],int counter){
  snprintf(iv,16,"%d",atoi(iv)+counter);
  printf("[--] New IV: ");
  for(int i = 0; i < 16; i++)
    printf("%02x",iv[i]);
  printf("\n");
}*/

void xor_string(unsigned char* str1, unsigned char* str2, unsigned char* result){
  for(int i = 0; i < 16; i++){
  //  printf("BL: %02x PL: %02x X: %02x\n",str1[i],str2[i],str1[i]^str2[i]);
    result[i] = str1[i] ^ str2[i];
  //  printf("ST[%d]: %02x\n",i,result[i]);
  }
}

void build_counters(unsigned char iv[16], int num_blocks,unsigned char (*counters)[16]){
  for(int i = 0; i < num_blocks; i++){
    for(int j = 0; j < 16; j++) counters[i][j] = iv[j];
    for(int j = 15; j > 0; j--){
      if(iv[j] == 255){iv[j]=0; continue;}
      iv[j]++;
      break;
    }
  }
}

void build_blocks(unsigned char* plain, int num_blocks, unsigned char (*blocks)[16],int size){
  for(int i = 0; i < num_blocks; i++){
    for(int j = 0; j < 16 ; j++){
      blocks[i][j]=plain[i*16+j];
    }
    //strncpy(&blocks[i][0],&plain[i*16],16);
  }
}

void print_counters(unsigned char (*counters)[16],int num_blocks){
  for(int i = 0; i < num_blocks; i++){
    printf("[--] Counter %d: ", i);
    for(int j = 0; j < 16; j++)
      printf("%02x",counters[i][j]);
    printf("\n");
  }
}


void print_blocks(unsigned char (*blocks)[16],int num_blocks){
  for(int i = 0; i < num_blocks; i++){
    printf("[--] Block %d: ", i);
    for(int j = 0; j < 16; j++)
      printf("%02x [%c]|",blocks[i][j],blocks[i][j]);
    printf("\n");
  }
}

int ctr_exec(unsigned char* plain, unsigned char* result, unsigned char* sub_keys, int rounds,int text_length){
  int num_blocks = (text_length / 16)+((text_length % 16)!=0);
	int tpb = 1024;

	if(sub_keys == NULL){
		printf("[xx] Error: you must pass the subkeys set!\n");
		exit(1);
	}

	// map message in gpu
	unsigned char* dev_plain;
	hipMalloc((void**)&dev_plain,text_length*sizeof(unsigned char));
	hipMemcpy(dev_plain,plain,text_length*sizeof(unsigned char),hipMemcpyHostToDevice);

	// map result zone in gpu
	unsigned char *dev_result;
	hipMalloc((void **)&dev_result, text_length * sizeof(unsigned char));
	hipMemcpy(dev_result, plain, text_length * sizeof(unsigned char), hipMemcpyHostToDevice);

	// map sbox on gpu
	unsigned char *dev_sbox;
	hipMalloc((void **)&dev_sbox, 256 * sizeof(unsigned char));
	hipMemcpy(dev_sbox, sbox, 256 * sizeof(unsigned char), hipMemcpyHostToDevice);

	//map subkeys on gpu
	unsigned char *dev_keys;
	hipMalloc((void **)&dev_keys, 10 * 16 * sizeof(unsigned char));
	hipMemcpy(dev_keys, sub_keys, 10 * 16 * sizeof(unsigned char), hipMemcpyHostToDevice);

	//execute aes
	int blck = ceil(num_blocks/tpb)==0?1:ceil(num_blocks/tpb);
  aes_encript<<<num_blocks , tpb>>>(dev_plain, dev_result, dev_sbox, dev_keys, text_length);

	//map result to main memory
	hipMemcpy(result, dev_result, text_length * sizeof(unsigned char), hipMemcpyDeviceToHost);
	result[text_length]=0x0;
//	printf("[--] RES: %s\n",result);
	//free cuda
	hipFree(dev_result);
	hipFree(dev_plain);
	hipFree(dev_keys);
	hipFree(dev_sbox);

	return num_blocks;
}
